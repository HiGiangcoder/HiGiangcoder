#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>

using namespace std;
using ll = long long;
const unsigned long long K = (1ULL << 63) | 13ULL;

__device__ long long simulate_device(long long s, long long n) {
    int v0[128];
    int v1[128];
    int len0 = 0, len1 = 0;

    double f = 0, na = 0, ng = 0;
    double a = -19;
    long long r = -19;
    const unsigned long long K = (1ULL << 63) | 13ULL;

    while (a < 1.0 * s * s) {
        na += 1.0 / 19.0;
        double f1 = f;

        // Sinh bit cho v0, v1
        for (int i2 = 0; i2 < 2; ++i2) {
            long long m = 19;
            int* v = (i2 == 0 ? v0 : v1);
            int& len = (i2 == 0 ? len0 : len1);
            len = 0;

            while (m) {
                m = n;
                len = 0;
                while (m > 0 && len < 128) {
                    int bit = (r & 1LL) ? 1 : 0;
                    v[len++] = bit;
                    long long rb = (r >> 1);
                    unsigned long long next = (unsigned long long)rb ^ (bit ? K : 0ULL);
                    r = (long long)next;
                    m -= 1 + bit;
                }
            }
        }

        bool e = true;
        int L = len0 < len1 ? len0 : len1;
        for (int i = 0; i < L; ++i) {
            if (v0[i] != v1[i]) { e = false; break; }
        }

        if (e) {
            ng += s;
            f = llround(ng / na);
            a = (f == f1) ? (a + 1.0 / 1919.0) : 0;
        }
    }
    return (long long)f;
}

__global__ void kernel_simulate(long long *s_arr, long long *n_arr, long long *out, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        out[idx] = simulate_device(s_arr[idx], n_arr[idx]);
    }
}

int main() {
    const int N = 10000;
    ll *h_s = new ll[N];
    ll *h_n = new ll[N];
    for (int i = 0; i < N; ++i) {
        h_s[i] = 1 + i / 100;
        h_n[i] = 1 + i % 100;
    }

    ll *d_s, *d_n, *d_out;
    hipMalloc(&d_s, N * sizeof(ll));
    hipMalloc(&d_n, N * sizeof(ll));
    hipMalloc(&d_out, N * sizeof(ll));

    hipMemcpy(d_s, h_s, N * sizeof(ll), hipMemcpyHostToDevice);
    hipMemcpy(d_n, h_n, N * sizeof(ll), hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (N + threads - 1) / threads;
    kernel_simulate<<<blocks, threads>>>(d_s, d_n, d_out, N);

    ll *h_out = new ll[N];
    hipMemcpy(h_out, d_out, N * sizeof(ll), hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; ++i) // in thử 10 kết quả đầu
        cout << h_s[i] << " " << h_n[i] << " -> " << h_out[i] << "\n";

    hipFree(d_s);
    hipFree(d_n);
    hipFree(d_out);
    delete[] h_s;
    delete[] h_n;
    delete[] h_out;

    return 0;
}